#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

extern "C" {
	#include "c_timer.h"
}

#define BLOCK_WIDTH 32

int main(int argc, char* argv[]) {
	float *A=NULL, *B=NULL, *C=NULL, *A_d=NULL, *B_d=NULL, *C_d=NULL;
	int i=0, N=0, M=0, P=0, nbrow=0, nbcol=0;
	double inizio, fine;

	__global__ void matmat(float *, float *, float *, int, int, int);
	void printMatrix(float *, const char[], int, int);

	if(argc == 6) {
		N = atoi(argv[1]);
		M = atoi(argv[2]);
		P = atoi(argv[3]);
		nbrow = atoi(argv[4]);
		nbcol = atoi(argv[5]);
	} else if(argc == 4) {
		N = atoi(argv[1]);
		M = atoi(argv[2]);
		P = atoi(argv[3]);
	} else {
		fprintf(stderr, " %s N M P [nbrow] [nbcol] -- [] means optional", argv[0]);
		perror(" ");
		return 0;
	}

	A = (float *) malloc(N * M * sizeof(float));
	B = (float *) malloc(M * P * sizeof(float));
	C = (float *) malloc(N * P * sizeof(float));

	hipMalloc((void **) &A_d, N * M * sizeof(float));
	hipMalloc((void **) &B_d, M * P * sizeof(float));
	hipMalloc((void **) &C_d, N * P * sizeof(float));

	for(i=0; i < N * M; i++)
		*(A + i) = i+1;
	for(i=0; i < M * P; i++)
		*(B + i) = i+1;
	for(i=0; i < N * P; i++)
		*(C + i) = 0;

	hipMemcpy(A_d, A, N * M * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(B_d, B, M * P * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(C_d, C, N * P * sizeof(float), hipMemcpyHostToDevice);

	dim3 DimGrid(1, 1);
	dim3 DimBlock(1, 1);

	if(argc == 4) {
		nbrow = ceil((double)N/BLOCK_WIDTH);
		nbcol = ceil((double)P/BLOCK_WIDTH);
		DimGrid.x = nbcol;
		DimGrid.y = nbrow;
		DimBlock.x = BLOCK_WIDTH;
		DimBlock.y = BLOCK_WIDTH;
	} else {
		nbrow += N%nbrow;
		nbcol += P%nbcol;
		DimGrid.x = nbcol;
		DimGrid.y = nbrow;
		DimBlock.x = P/nbcol;
		DimBlock.y = N/nbrow;
	}

	inizio = get_cur_time();
	matmat<<<DimGrid, DimBlock>>>(A_d, B_d, C_d, N, M, P);
	hipDeviceSynchronize();
	fine = get_cur_time();
	printf(" GPU Computation Time: %lfs\n", fine - inizio);
	printf(" GPU Gflop/s: %e\n", ((double)2*M*N*P)/(fine - inizio));
	printf("\n");

	hipMemcpy(C, C_d, N * P * sizeof(float), hipMemcpyDeviceToHost);

	/*printMatrix(A, "A", N, M);
	printf("\n");
	printMatrix(B, "B", M, P);
	printf("\n");
	printMatrix(C, "C", N, P);
	printf("\n");*/

	free(A);
	free(B);
	free(C);
	hipFree(A_d);
	hipFree(B_d);
	hipFree(C_d);

	return 0;
}

__global__ void matmat(float *A_d, float *B_d, float *C_d, int N, int M, int P) {
	int k=0, row = blockIdx.y * blockDim.y + threadIdx.y, col = blockIdx.x * blockDim.x + threadIdx.x;

	if(row < N && col < P) {
		for(k=0; k < M; k++)
			C_d[row * P + col] += A_d[row * M + k] * B_d[k * P + col];
		//printf(" %d|%d %d|%d C(%d,%d):%.2f\n", blockIdx.y, blockIdx.x, threadIdx.y, threadIdx.x, row, col, C_d[row * P + col]);
	}
}

void printMatrix(float *M, const char name[], int row, int col) {
	int i;
	printf(" %s:", name);
	for(i=0; i < row * col; i++) {
		if(i % col == 0) printf("\n");
		printf("  %.2f ", *(M + i));
	}
	printf("\n");
}
